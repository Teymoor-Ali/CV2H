#include <iostream>
#include <opencv2/imgproc.hpp>
#include "FFT.h"
#include "STREAM.cuh"
#include <hip/hip_runtime.h>
#include ""

using namespace cv;

int main(int argc, char* argv[]) {
	if (argc < 2) {
		std::cout << "Usage: " << argv[0] << " <function> [args...]" << std::endl;
		return 1;
	}

	std::string function = argv[1];

	if (function == "FFT") {
		if (argc != 3) {
			std::cout << "Usage: " << argv[0] << " FFT <size>" << std::endl;
			return 1;
		}

		int size = std::stoi(argv[2]);
		double cpu_time, gpu_time;
		int cpu_mem_usage, gpu_mem_usage;
		fft(size, size, cpu_time, cpu_mem_usage, gpu_time, gpu_mem_usage);

		std::cout << "CPU time: " << cpu_time << " seconds" << std::endl;
		std::cout << "CPU memory usage: " << cpu_mem_usage << " MB" << std::endl;
		std::cout << "GPU time: " << gpu_time << " seconds" << std::endl;
		std::cout << "GPU memory usage: " << gpu_mem_usage << " MB" << std::endl;
	}
	else if (function == "STREAM") {
		if (argc != 3) {
			std::cout << "Usage: " << argv[0] << " STREAM <num_streams>" << std::endl;
			return 1;
		}

		int num_streams = std::stoi(argv[2]);
		STREAM(num_streams);
	}
	else {
		std::cout << "Unknown function: " << function << std::endl;
		return 1;
	}

	return 0;
}
